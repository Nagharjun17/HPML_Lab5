#include "hip/hip_runtime.h"
%%cu
#include<iostream>
#include<math.h>
#include<ctime>

__global__ 
void add(int n, float *x, float*y)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = blockDim.x*gridDim.x;
    for(int i=index; i<n; i+=stride)
    {
        y[i] = x[i] + y[i];
    }
}

int main(void)
{
    int K = 1<<20;
    
    float *x, *y;
    int count = 1;
    int arr[] = {1,5,2,5,2};
    
    for(int i=0;i<5;i++)
    {
    
    K = K*arr[i];
    count = count*arr[i];
     
    size_t size = K*sizeof(float);
    float *hx = (float*)malloc(size);
    float *hy = (float*)malloc(size);
 
    for(int i=0;i<K;i++)
    {
        hx[i] = 2.0f;
        hy[i] = 3.0f;
    }
 
    hipMalloc(&x, size);   
    hipMalloc(&y, size);
 
    clock_t start = clock();

    hipMemcpy(x, hx, size, hipMemcpyHostToDevice);
    hipMemcpy(y, hy, size, hipMemcpyHostToDevice);
     
    add<<<1,1>>>(K,x,y);

    hipMemcpy(hy, y, size, hipMemcpyDeviceToHost);
 
    clock_t end = clock();
    double time_taken = double(end - start) / double(CLOCKS_PER_SEC);
 
    std::cout<<"Time taken for "<<count<<"M elements is : "<<time_taken<<std::endl;
     
    free(hx);
    free(hy);
    }
    hipFree(x);
    hipFree(y);
    
    return 0;

}
